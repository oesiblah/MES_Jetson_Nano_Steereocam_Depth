#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <iomanip>
#include <sys/time.h>
#include <string>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/ximgproc.hpp>

#include "TaraXL.h"
#include "TaraXLCam.h"
#include "TaraXLDepth.h"

#include "detectNet.h"
#include <signal.h>
#include "cudaRGB.h"

using namespace std;
using namespace cv;
using namespace TaraXLSDK;

int main (int argc, char** argv) {

    commandLine cmdLine(argc, argv, (const char*)NULL);
    TaraXL taraxlCam;
    TaraXLCamList taraxlCamList;
    ResolutionList supportedResolutions;
    ACCURACY_MODE selectedMode;

    uint iAccuracyMode;
    TARAXL_STATUS_CODE status;

    detectNet* net = detectNet::Create(cmdLine);
    if(!net)
    {
        LogError("detectNet: failed to load detectNet model\n");
        return 0;
    }

    const uint32_t overlayFlags = detectNet::OverlayFlagsFromStr("box,labels,conf");

    status = taraxlCam.enumerateDevices(taraxlCamList);
    if (status != TARAXL_SUCCESS) {

        cout << "Camera enumeration failed" << endl;
        return 1;
    }

    if (taraxlCamList.size() == 0) {

        cout << "No cameras connected" << endl;
        return 1;
    }
    cout << endl << "Select a Accuracy mode:" << endl;
    cout << "0: High Accuracy" << endl <<"1: Low Accuracy "<<endl<<"2: Ultra Accuracy" << endl;
    cin >> iAccuracyMode;

    if (cin.fail()) {

        cout << "Invalid input" << endl;
        return 1;
    }

    if (iAccuracyMode == 0) {

        selectedMode = HIGH;
    }
    else if (iAccuracyMode == 1) {

        selectedMode = LOW;
    }
    else if (iAccuracyMode == 2) {

        selectedMode = ULTRA;
    }

    else {

        cout << "Invalid input" << endl;
        return 1;
    }

    vector<Ptr<TaraXLDepth> > taraxlDepthList;
    vector<Mat> left, right, grayDisp, colorDisp, depthMap;
    vector<string> cameraUniqueIdList;
    Resolution res;
    for(int i = 0 ; i < taraxlCamList.size() ; i++)
    {
        status = taraxlCamList.at(i).connect();
        if (status != TARAXL_SUCCESS) {

            cout << "Camera connect failed " << status << endl;
            return 1;
        }
        Ptr<TaraXLDepth> depth;
        cout << "Camera connect status" << status << endl;
        depth = new TaraXLDepth(taraxlCamList.at(i));
        if (depth == NULL)
        {
            cout << "Unable to create instance to TaraDepth" << endl;
            return 1;
        }
        depth->setAccuracy(selectedMode);
        taraxlDepthList.push_back(depth);
            string id;
            taraxlCamList.at(i).getCameraUniqueId(id);
        cameraUniqueIdList.push_back(id);
        string windowName = "CAMERA : "+ id;
        namedWindow(windowName, CV_WINDOW_AUTOSIZE);

        Mat sample;
        left.push_back(sample);
        right.push_back(sample);
        grayDisp.push_back(sample);
        colorDisp.push_back(sample);
        depthMap.push_back(sample);
        taraxlCamList.at(i).getResolution(res);
    }
    
    while(1)
    {
        for(int i = 0 ; i < taraxlCamList.size() ; i++)
        {
            status = taraxlDepthList.at(i)->getMap(left.at(i), right.at(i), grayDisp.at(i), true, depthMap.at(i), true, TARAXL_DEFAULT_FILTER);
            if (status != TARAXL_SUCCESS)
            {
                cout << "Get map failed" << endl;
                delete taraxlDepthList.at(i);
                return 1;
            }
            grayDisp.at(i).convertTo(grayDisp.at(i),CV_8U);

            detectNet::Detection* detections = NULL;
            Mat left_color;
            cvtColor(left.at(i), left_color, COLOR_GRAY2BGR);

            uchar3* imgBufferRGB = NULL;
            float4* imgBufferRGBAf = NULL;
            hipMalloc((void**)&imgBufferRGB, left_color.cols * sizeof(uchar3) * left_color.rows);
            hipMalloc((void**)&imgBufferRGBAf, left_color.cols * sizeof(float4) * left_color.rows);
            hipMemcpy2D((void*)imgBufferRGB, left_color.cols*sizeof(uchar3), (void*)left_color.data, left_color.step, left_color.cols*sizeof(uchar3), left_color.rows, hipMemcpyHostToDevice);
            cudaRGB8ToRGBA32(imgBufferRGB, imgBufferRGBAf, left_color.cols, left_color.rows);

            const int numDetections = net->Detect((float*)imgBufferRGBAf, (uint32_t)left_color.cols, (uint32_t)left_color.rows, &detections, overlayFlags);

            cv::cvtColor(left.at(i), left.at(i), COLOR_GRAY2BGR);
            if(numDetections > 0)
            {
                for(int n=0; n < numDetections; n++)
                {
                    LogVerbose("detected obj %i class #%u (%s) confidence=%f\n", n, detections[n].ClassID, net->GetClassDesc(detections[n].ClassID), detections[n].Confidence);
                    LogVerbose("bounding box %i  (%f, %f)  (%f, %f)  w=%f  h=%f\n", n, detections[n].Left, detections[n].Top, detections[n].Right, detections[n].Bottom, detections[n].Width(), detections[n].Height());
                    Point p1(detections[n].Left + detections[n].Width(), detections[n].Top + detections[n].Height());
                    Point p2(detections[n].Left, detections[n].Top);
                    Scalar magenta = Scalar(255,10,255);
                    Point center(detections[n].Left + detections[n].Width()*0.5, detections[n].Top + detections[n].Height()*0.5);
                    putText(left.at(i), net->GetClassDesc(detections[n].ClassID), Point(detections[n].Left + detections[n].Width()*0.5, (detections[n].Top + 25)), FONT_HERSHEY_SIMPLEX, 1, Scalar(0,200,200), 2);
                    rectangle(left.at(i), p1, p2, magenta, 1, 8, 0);

                    Mat disp, disp_32;
                    float DepthValue;
                    Rect recROI(detections[n].Left + detections[n].Width()*0.5, detections[n].Top + detections[n].Height()*0.5, 1, 1);
                    disp = depthMap.at(i)(recROI);
                    disp.convertTo(disp_32, CV_32FC1, 1.0);
                    Scalar MeanDisp = mean(disp_32);
                    DepthValue = (float)MeanDisp.val[0];
                    String test;
                    test = "Distance: " + std::to_string(DepthValue) + "cm";
                    putText(left.at(i), test, Point(detections[n].Left + detections[n].Width()*0.5, (detections[n].Top + 50)), FONT_HERSHEY_SIMPLEX, 1, Scalar(0,200,200), 2);
                    std::cout << "Object " << n << " distance: " << DepthValue << "cm" << std::endl;
                }
            }

            string windowName = "CAMERA : "+ cameraUniqueIdList.at(i);
            imshow(windowName, left.at(i));
            hipFree(imgBufferRGB);
            hipFree(imgBufferRGBAf);
        }
        int keycode = waitKey(30) & 0xff;
        if(keycode == 27) break;
    }
    exit(0);
}
